#include "common.h"
#include <chrono>
#include <cmath>
#include <cstring>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <random>
#include <vector>

// =================
// Helper Functions
// =================

// I/O routines
void save(std::ofstream& fsave, particle_t* parts, int num_parts, double size) {
    static bool first = true;

    if (first) {
        fsave << num_parts << " " << size << std::endl;
        first = false;
    }

    for (int i = 0; i < num_parts; ++i) {
        fsave << parts[i].x << " " << parts[i].y << std::endl;
    }

    fsave << std::endl;
}


// Command Line Option Processing
int find_arg_idx(int argc, char** argv, const char* option) {
    for (int i = 1; i < argc; ++i) {
        if (strcmp(argv[i], option) == 0) {
            return i;
        }
    }
    return -1;
}

int find_int_arg(int argc, char** argv, const char* option, int default_value) {
    int iplace = find_arg_idx(argc, argv, option);

    if (iplace >= 0 && iplace < argc - 1) {
        return std::stoi(argv[iplace + 1]);
    }

    return default_value;
}

char* find_string_option(int argc, char** argv, const char* option, char* default_value) {
    int iplace = find_arg_idx(argc, argv, option);

    if (iplace >= 0 && iplace < argc - 1) {
        return argv[iplace + 1];
    }

    return default_value;
}

// ==============
// Main Function
// ==============

int main(int argc, char** argv) {

    bool debug = true;
  
    // check that a file name is specified
    if (argc != 2) {
        cout << "Need to specify 1 file to read edge list from" << '\n';
        return 0;
    }
    // initialize graph variables
    int num_edges = 0;
    int num_vertices = 0;
    edge_t* edges;

    // Citation from https://www.cplusplus.com/doc/tutorial/files/ for reading from a file
    // read edge list from text file, assuming well formatted text file
    // first line of file: num_vertices num_edges
    // rest of lines: vertex_1 vertex_2 edge_weight
    string line;
    ifstream myfile (argv[1]);
    int line_cnt = 0;
    if (myfile.is_open())
      {
        while ( getline (myfile, line) )
        {
          line_cnt += 1;
          if (line_cnt > num_edges + 1) {
            cout << "File incorrectly formatted, too many edges" << '\n';
            myfile.close();
            return 0;
          }
          if (line_cnt == 1) {
              // Citation from https://www.javatpoint.com/how-to-split-strings-in-cpp
              // for parsing and splitting strings
              char* curr_ptr = strtok(line, " ");
              if (curr_ptr == NULL) {
                cout << "File incorrectly formatted, no num_vertices given" << '\n';
                myfile.close();
                return 0;
              }
              num_vertices = atoi(curr_ptr);
              curr_ptr = strtok(NULL, " ");
              if (curr_ptr == NULL) {
                cout << "File incorrectly formatted, no num_edges given" << '\n';
                myfile.close();
                return 0;
              }
              num_edges = atoi(curr_ptr);
              curr_ptr = strtok(NULL, " ");
              if (curr_ptr != NULL) {
                cout << "File incorrectly formatted" << '\n';
                myfile.close();
                return 0;
              }
              if (num_edges == 0) {
                cout << "No edge graph, return" << '\n';
                return 0;
              }
              // allocate array for edges
              edges = new edge_t[num_edges];
          } else {
              // Citation from https://www.javatpoint.com/how-to-split-strings-in-cpp
              // for parsing and splitting strings
              int edge_posn = line_cnt - 2;
              char* curr_ptr = strtok(line, " ");
              if (curr_ptr == NULL) {
                cout << "File incorrectly formatted, no vertex_1 given" << '\n';
                myfile.close();
                return 0;
              }
              edges[edge_posn].vertex_1 = atoi(curr_ptr);
              curr_ptr = strtok(NULL, " ");
              if (curr_ptr == NULL) {
                cout << "File incorrectly formatted, no vertex_2 given" << '\n';
                myfile.close();
                return 0;
              }
              edges[edge_posn].vertex_2 = atoi(curr_ptr);
              if (curr_ptr == NULL) {
                cout << "File incorrectly formatted, no vertex_2 given" << '\n';
                myfile.close();
                return 0;
              }
              edges[edge_posn].weight = atof(curr_ptr);
              if (curr_ptr != NULL) {
                cout << "File incorrectly formatted" << '\n';
                myfile.close();
                return 0;
              }
          }
        }
        myfile.close();
        if (line_cnt != num_edges + 1) {
            cout << "File incorrectly formatted, too few edges specified" << '\n';
            return 0;
        }
    }

    else 
    {
        cout << "Unable to open file";
        return 0;
    }
    if (debug) {
        cout << "Num Vertices: " << num_vertices << " Num Edges: " << num_edges << '\n';
        for (int i = 0; i < num_edges; i++) {
            cout << "Edge: " << i << " vertex_1: " << edges[i].vertex_1 << " vertex_2: " << edges[i].vertex_2 << " weight: " << edges[i].weight << '\n';
        }
    }
    free(edges);
}
